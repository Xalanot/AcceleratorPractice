#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <chrono>
#include <thrust/memory.h>
#include <thrust/system/cuda/memory.h>

#include "common_pinned_memory.h"

#define DEBUG 1

class fallback_allocator
{
  public:
    // just allocate bytes
    typedef char value_type;

    // allocate's job to is allocate host memory as a functional fallback when hipMalloc fails
    char *allocate(std::ptrdiff_t n)
    {
      char *result = 0;

      // attempt to allocate device memory
      if(hipMalloc(&result, n) == hipSuccess)
      {
        if(DEBUG)
            std::cout << "  allocated " << n << " bytes of device memory" << std::endl;
      }
      else
      {
        // reset the last CUDA error
        hipGetLastError();

        // attempt to allocate pinned host memory
        void *h_ptr = 0;
        if(hipHostMalloc(&h_ptr, n) == hipSuccess)
        {
          // attempt to map host pointer into device memory space
          if(hipHostGetDevicePointer(&result, h_ptr, 0) == hipSuccess)
          {
            if(DEBUG)
                std::cout << "  allocated " << n << " bytes of pinned host memory (fallback successful)" << std::endl;
          }
          else
          {
            // reset the last CUDA error
            hipGetLastError();

            // attempt to deallocate buffer
            if(DEBUG)
                std::cout << "  failed to map host memory into device address space (fallback failed)" << std::endl;
            hipHostFree(h_ptr);

            throw std::bad_alloc();
          }
        }
        else
        {
          // reset the last CUDA error
          hipGetLastError();
          if(DEBUG)
            std::cout << "  failed to allocate " << n << " bytes of memory (fallback failed)" << std::endl;

          throw std::bad_alloc();
        }
      }

      return result;
    }

    // deallocate's job to is inspect where the pointer lives and free it appropriately
    void deallocate(char *ptr, size_t n)
    {
      void *raw_ptr = thrust::raw_pointer_cast(ptr);

      // determine where memory resides
      hipPointerAttribute_t	attributes;

      if(hipPointerGetAttributes(&attributes, raw_ptr) == hipSuccess)
      {
        // free the memory in the appropriate way
        if(attributes.memoryType == hipMemoryTypeHost)
        {
          hipHostFree(raw_ptr);
        }
        else
        {
          hipFree(raw_ptr);
        }
      }
    }
};

void sort1(size_t numberOfElements)
{
    size_t memSize = sizeof(int) * numberOfElements;
    checkDevice(memSize);

    int* hostMemPointer = nullptr;
    checkCudaError(hipHostAlloc((void**)&hostMemPointer, memSize, 0));

    thrust::tabulate(hostMemPointer, hostMemPointer + numberOfElements, get_rand_number(1337, 10 * numberOfElements));
    
    // copy to device with hostpointer
    thrust::device_vector<int> device_vec(hostMemPointer, hostMemPointer + numberOfElements);
    // sort on device
    thrust::sort(device_vec.begin(), device_vec.end());
    // transfer back to host
    thrust::host_vector<int> host_vec = device_vec;

    hipHostFree(hostMemPointer);
}

void sort2(size_t numberOfElements)
{
    size_t memSize = sizeof(int) * numberOfElements;
    checkDevice(memSize);

    int* hostMemPointer = nullptr;
    checkCudaError(hipHostAlloc((void**)&hostMemPointer, memSize, hipHostMallocPortable));

    thrust::tabulate(hostMemPointer, hostMemPointer + numberOfElements, get_rand_number(1337, 10 * numberOfElements));
    
    // copy to device with hostpointer
    thrust::device_vector<int> device_vec(hostMemPointer, hostMemPointer + numberOfElements);
    // sort on device
    thrust::sort(device_vec.begin(), device_vec.end());
    // transfer back to host
    thrust::host_vector<int> host_vec = device_vec;

    hipHostFree(hostMemPointer);   
}

void sort3(size_t numberOfElements)
{
    fallback_allocator alloc;

    // use our special malloc to allocate
    int *raw_ptr = reinterpret_cast<int*>(alloc.allocate(numberOfElements * sizeof(int)));        

    thrust::cuda::pointer<int> begin = thrust::cuda::pointer<int>(raw_ptr);
    thrust::cuda::pointer<int> end   = begin + numberOfElements;        

    thrust::tabulate(begin, end, get_rand_number(1337, numberOfElements));
    try{
        thrust::sort(thrust::cuda::par(alloc), begin, end);
    }
    catch(std::bad_alloc){
        std::cout << "  caught std::bad_alloc from thrust::sort" << std::endl;
    }

    alloc.deallocate(reinterpret_cast<char*>(raw_ptr), vecSize * sizeof(int));
}

int main(int argc, char *argv[]){
    size_t numberOfElements= static_cast<size_t>(1) << 2;
    sort3(numberOfElements);

    return 0;
}