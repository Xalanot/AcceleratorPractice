#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <chrono>
#include <thrust/memory.h>
#include <thrust/system/cuda/memory.h>

#include "cached_allocator.h"
#include "common_pinned_memory.h"
#include "fallback_allocator.h"

#include "../pml/csvwriter.h"
#include "../pml/measurement.h"

#define DEBUG 0

template<typename T>
void sort1(size_t numberOfElements, MeasurementSeries<T>& measurementSeries)
{
    size_t memSize = sizeof(int) * numberOfElements;
    checkDevice(memSize);

    int* hostMemPointer = nullptr;
    checkCudaError(hipHostAlloc((void**)&hostMemPointer, memSize, 0));

    thrust::tabulate(hostMemPointer, hostMemPointer + numberOfElements, get_rand_number(1337, 10 * numberOfElements));
    
    hipDeviceSynchronize();
    measurementSeries.start();
    // copy to device with hostpointer
    thrust::device_vector<int> device_vec(hostMemPointer, hostMemPointer + numberOfElements);
    // sort on device
    thrust::sort(device_vec.begin(), device_vec.end());
    // transfer back to host
    thrust::host_vector<int> host_vec = device_vec;
    hipDeviceSynchronize();
    measurementSeries.stop();

    hipHostFree(hostMemPointer);
}

template<typename T>
void sort2(size_t numberOfElements, MeasurementSeries<T>& measurementSeries)
{
    size_t memSize = sizeof(int) * numberOfElements;
    checkDevice(memSize);

    int* hostMemPointer = nullptr;
    checkCudaError(hipHostAlloc((void**)&hostMemPointer, memSize, hipHostMallocPortable));

    thrust::tabulate(hostMemPointer, hostMemPointer + numberOfElements, get_rand_number(1337, 10 * numberOfElements));
    
    thrust::device_ptr<int> ptr = thrust::device_pointer_cast(hostMemPointer);
    hipDeviceSynchronize();
    measurementSeries.start();
    thrust::sort(ptr, ptr + numberOfElements);
    hipDeviceSynchronize();
    measurementSeries.stop();
    assert(thrust::is_sorted(ptr, ptr + numberOfElements));

    hipHostFree(hostMemPointer);   
}

template<typename T>
void sort3(size_t numberOfElements, MeasurementSeries<T>& measurementSeries)
{
    fallback_allocator alloc;
    std::cout << "sort3" << std::endl;

    // use our special malloc to allocate
    int *raw_ptr = reinterpret_cast<int*>(alloc.allocate(numberOfElements * sizeof(int)));        

    thrust::cuda::pointer<int> begin = thrust::cuda::pointer<int>(raw_ptr);
    thrust::cuda::pointer<int> end   = begin + numberOfElements;        

    thrust::tabulate(begin, end, get_rand_number(1337, numberOfElements));
    hipDeviceSynchronize();
    measurementSeries.start();
    try{
        thrust::sort(thrust::cuda::par(alloc), begin, end);
    }
    catch(std::bad_alloc){
        std::cout << "  caught std::bad_alloc from thrust::sort" << std::endl;
    }

    hipDeviceSynchronize();
    measurementSeries.stop();

    alloc.deallocate(reinterpret_cast<char*>(raw_ptr), numberOfElements * sizeof(int));
}

template <typename T>
void sort4(size_t numberOfElements, MeasurementSeries<T>& measurementSeries)
{
  size_t int_size = sizeof(int);
  if (checkDevice(numberOfElements * int_size))
  {
    return sort3(numberOfElements, measurementSeries);
  }

  thrust::host_vector<int> X_h(numberOfElements);
  thrust::tabulate(X_h.begin(), X_h.end(), get_rand_number(1337, 10 * numberOfElements));

  std::vector<hipStream_t> streams(3);
  for (auto& stream : streams)
  {
    hipStreamCreate(&stream);
  }

  std::vector<size_t> sizes(3);
  for (size_t i = 0; i < sizes.size(); ++i)
  {
    if (i != sizes.size() - 1)
    {
      sizes[i] = numberOfElements / sizes.size();
    }
    else
    {
      sizes[i] = (numberOfElements + sizes.size() - 1) / sizes.size();
    }

    std::cout << sizes[i] << std::endl;
  }

  #pragma omp parallel for num_threads(deviceCount) shared(result)
  for (int i = 0; i < 3; ++i)
  {
    thrust::device_vector<int> X_d(sizes[i]);
    checkCudaError(hipMemcpyAsync(thrust::raw_pointer_cast(X_d.data()), thrust::raw_pointer_cast(X_h.data() + i * sizes[0]), sizes[i] * int_size, hipMemcpyHostToDevice, streams[i]));
    cached_allocator allocator;
    thrust::sort(thrust::cuda::par(allocator).on(streams[i]), X_d.begin(), X_d.end());
    assert(thrust::is_sorted(X_d.begin(), X_d.end()));
  }
  
}

int main(int argc, char *argv[]){
    int iterations = 1;

    std::vector<int> sizes;
    std::vector<MeasurementSeries<std::chrono::milliseconds>> sort1Times;
    std::vector<MeasurementSeries<std::chrono::milliseconds>> sort2Times;
    std::vector<MeasurementSeries<std::chrono::milliseconds>> sort3Times;

    size_t N = static_cast<size_t>(1) << 31;
    MeasurementSeries<std::chrono::milliseconds> sort3Series;
    sort4(N, sort3Series);


    /*for (size_t i = 20; i < 25; ++i)
    {
        std::cout << i << std::endl;
        size_t numberOfElements = static_cast<size_t>(1) << i;
        if (!checkDevice(sizeof(int) * i))
        {
            break;
        }

        sizes.push_back(i);

        MeasurementSeries<std::chrono::milliseconds> sort1Series;
        for (int j = 0; j < iterations; ++j)
        {
            sort1(numberOfElements, sort1Series);
        }
        sort1Times.push_back(sort1Series);

        MeasurementSeries<std::chrono::milliseconds> sort2Series;
        for (int j = 0; j < iterations; ++j)
        {
            sort2(numberOfElements, sort2Series);
        }
        sort2Times.push_back(sort2Series);

        MeasurementSeries<std::chrono::milliseconds> sort3Series;
        for (int j = 0; j < iterations; ++j)
        {
            sort3(numberOfElements, sort3Series);
        }
        sort3Times.push_back(sort3Series);
    }

    CSVWriter csvwriter("pinned_memory.csv");
    csvwriter.setHeaderNames( {"size", "sort1", "sort2", "sort3"});

    csvwriter.write(sizes, sort1Times, sort2Times, sort3Times);*/

    return 0;
}