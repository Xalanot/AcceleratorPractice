#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <chrono>
#include <thrust/memory.h>
#include <thrust/system/cuda/memory.h>

#include "common_pinned_memory.h"


class fallback_allocator
{
  public:
    // just allocate bytes
    typedef char value_type;

    // allocate's job to is allocate host memory as a functional fallback when hipMalloc fails
    char *allocate(std::ptrdiff_t n)
    {
      char *result = 0;

      // attempt to allocate device memory
      if(hipMalloc(&result, n) == hipSuccess)
      {
        if(DEBUG)
            std::cout << "  allocated " << n << " bytes of device memory" << std::endl;
      }
      else
      {
        // reset the last CUDA error
        hipGetLastError();

        // attempt to allocate pinned host memory
        void *h_ptr = 0;
        if(hipHostMalloc(&h_ptr, n) == hipSuccess)
        {
          // attempt to map host pointer into device memory space
          if(hipHostGetDevicePointer(&result, h_ptr, 0) == hipSuccess)
          {
            if(DEBUG)
                std::cout << "  allocated " << n << " bytes of pinned host memory (fallback successful)" << std::endl;
          }
          else
          {
            // reset the last CUDA error
            hipGetLastError();

            // attempt to deallocate buffer
            if(DEBUG)
                std::cout << "  failed to map host memory into device address space (fallback failed)" << std::endl;
            hipHostFree(h_ptr);

            throw std::bad_alloc();
          }
        }
        else
        {
          // reset the last CUDA error
          hipGetLastError();
          if(DEBUG)
            std::cout << "  failed to allocate " << n << " bytes of memory (fallback failed)" << std::endl;

          throw std::bad_alloc();
        }
      }

      return result;
    }

    // deallocate's job to is inspect where the pointer lives and free it appropriately
    void deallocate(char *ptr, size_t n)
    {
      void *raw_ptr = thrust::raw_pointer_cast(ptr);

      // determine where memory resides
      hipPointerAttribute_t	attributes;

      if(hipPointerGetAttributes(&attributes, raw_ptr) == hipSuccess)
      {
        // free the memory in the appropriate way
        if(attributes.memoryType == hipMemoryTypeHost)
        {
          hipHostFree(raw_ptr);
        }
        else
        {
          hipFree(raw_ptr);
        }
      }
    }
};

void sort1(size_t numberOfElements)
{
    size_t memSize = sizeof(int) * numberOfElements;
    checkDevice(memSize);

    int* hostMemPointer = nullptr;
    checkCudaError(hipHostAlloc((void**)&hostMemPointer, memSize, 0));

    thrust::tabulate(hostMemPointer, hostMemPointer + numberOfElements, get_rand_number(1337, vecSize));
    
    // copy to device with hostpointer
    thrust::device_vector<int> device_vec(hostMemPointer, hostMemPointer + numberOfElements);
    for (int i = 0; i < device_vec.size(); ++i)
    {
        std::cout << device_vec[i] << std::endl;
    }
    std::cout << "sort" << std::endl
    // sort on device
    thrust::sort(device_vec.begin(), device_vec.end());
    // transfer back to host
    for (int i = 0; i < device_vec.size(); ++i)
    {
        std::cout << device_vec[i] << std::endl;
    }
    thrust::host_vector<int> host_vec = device_vec;


    hipHostFree(hostMemPointer);
}

int main(int argc, char *argv[]){
    size_t numberOfElements= static_cast<size_t>(1) << 2;
    sort1(numberOfElements);
    /*size_t vecSize;
    vecSize = atoll(argv[1]);
    size_t memSize = sizeof(int)*vecSize;
    int *hostMemPointer = NULL;
    double timer0, timer1, timer2;
    int device;
    int sufficientMemSize = 1;
    //int *deviceMemPointer = NULL;

    // 4.1.1 sort on gpu with copy from host and transfer back
    if(sufficientMemSize)
    {
        checkCudaError(hipHostAlloc((void**)&hostMemPointer, memSize, 0));

        thrust::tabulate(hostMemPointer, hostMemPointer + vecSize, get_rand_number(123, vecSize));

        hipDeviceSynchronize();
        auto timer_start = std::chrono::high_resolution_clock::now();
        
        // copy to device with hostpointer
        thrust::device_vector<int>device_vec(hostMemPointer, hostMemPointer + vecSize);
        // sort on device
        thrust::sort(device_vec.begin(), device_vec.end());
        // transfer back to host
        thrust::host_vector<int>host_vec = device_vec;

        auto timer_end = std::chrono::high_resolution_clock::now();
        timer0 = std::chrono::duration<double>(timer_end - timer_start).count();

        hipHostFree(hostMemPointer);
    }

    // 4.1.2 sort on gpu with no copy
    if(sufficientMemSize)
    {
        hostMemPointer = NULL;
        // allocate space on host in cuda adress space
        checkCudaError(hipHostAlloc((void**)&hostMemPointer, memSize, hipHostMallocPortable));

        thrust::tabulate(hostMemPointer, hostMemPointer + vecSize, get_rand_number(123, vecSize));
        
        // set device vector to pointer of host memory in cuda adress space
        thrust::device_vector<int>device_vec_unf(hostMemPointer, hostMemPointer + vecSize);
        
        hipDeviceSynchronize();
        auto timer_start = std::chrono::high_resolution_clock::now();
        // sort on device
        thrust::sort(device_vec_unf.begin(), device_vec_unf.end());
        auto timer_end = std::chrono::high_resolution_clock::now();

        timer1 = std::chrono::duration<double>(timer_end - timer_start).count();

        hipHostFree(hostMemPointer);
    }

    // 4.1.3 sort on host and device memory with fallback allocator
    {
        fallback_allocator alloc;

        // use our special malloc to allocate
        int *raw_ptr = reinterpret_cast<int*>(alloc.allocate(vecSize * sizeof(int)));        

        thrust::cuda::pointer<int> begin = thrust::cuda::pointer<int>(raw_ptr);
        thrust::cuda::pointer<int> end   = begin + vecSize;        

        thrust::tabulate(begin, end, get_rand_number(123, vecSize));

        hipDeviceSynchronize();
        auto timer_start = std::chrono::high_resolution_clock::now();
        try{
            thrust::sort(thrust::cuda::par(alloc), begin, end);
        }
        catch(std::bad_alloc){
            std::cout << "  caught std::bad_alloc from thrust::sort" << std::endl;
        }
        auto timer_end = std::chrono::high_resolution_clock::now();

        timer2 = std::chrono::duration<double>(timer_end - timer_start).count();

        alloc.deallocate(reinterpret_cast<char*>(raw_ptr), vecSize * sizeof(int));
    }

    std::cout << vecSize << "\t" << timer0 << "\t" << timer1 << "\t" << timer2 << "\t" << std::endl; */

    return 0;
}