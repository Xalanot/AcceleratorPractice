#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <iostream>
#include <iterator>
#include <algorithm>
#include <thrust/random.h>
#include <hip/hip_runtime.h>
#include <thrust/execution_policy.h>
#include <chrono>
#include <omp.h>
#include <cstdlib>
#include <iostream>
#include <sstream>
#include <map>
#include <cassert>
#include <thrust/system/cuda/memory.h>

// Error handeling of cuda functions
#define checkCudaError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
};

int main(int argc, char **argv)
{
    int deviceCount;
    checkCudaError(hipGetDeviceCount(&deviceCount));
    std::cout << "device count: " << deviceCount << std::endl;
}
