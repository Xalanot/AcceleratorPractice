#include "hip/hip_runtime.h"
#include "saxpy_multi_gpu.h"

template <typename T>
struct square
{
    __host__ __device__
        T operator()(const T& x) const { 
            return x * x;
        }
};

float norm_multi(float *X_h, size_t N, int deviceCount)
{    
    std::vector<DeviceManager> deviceManagers;
    for (int i = 0; i < deviceCount; ++i)
    {
        deviceManagers.emplace_back( DeviceManager{i} );
    }

    size_t deviceSize = N / deviceCount;
    size_t float_size = sizeof(float);
   
    float result = 0;
    square<float> unary_op;
    thrust::plus<float> binary_op;

    #pragma omp parallel for num_threads(deviceCount) shared(result)
    for(int i = 0; i < deviceCount; ++i){

        checkCudaError(hipSetDevice(i));
        
        thrust::device_vector<float>X_d(deviceSize);
        checkCudaError(hipMemcpyAsync(thrust::raw_pointer_cast(X_d.data()), X_h + i * deviceCount, deviceCount * float_size, hipMemcpyDefault, deviceManagers[i].h2dStream));

        // wait for copy to complete
        checkCudaError(hipEventRecord(deviceManagers[i].copyEvent, deviceManagers[i].h2dStream));
        hipStreamWaitEvent(deviceManagers[i].h2dStream, deviceManagers[i].copyEvent, 0);

        checkCudaError(hipEventRecord(deviceManagers[i].start, deviceManagers[i].transformStream));
        result += thrust::transform_reduce(thrust::cuda::par.on(deviceManagers[i].transformStream), X_d.begin(), X_d.end(), unary_op, 0, binary_op);
        checkCudaError(hipEventRecord(deviceManagers[i].stop, deviceManagers[i].transformStream));

        /*
        checkCudaError(hipEventSynchronize(myDevices[i].stop));
        checkCudaError(hipEventElapsedTime(&myDevices[i].myTime, myDevices[i].start, myDevices[i].stop));
        */
    }
    hipDeviceSynchronize();

    return sqrt(result);
}

float norm_single(float *X_h, size_t N, int deviceCount)
{
    size_t float_size = sizeof(float);
    thrust::device_vector<float>X_d(N);
    checkCudaError(hipMemcpy(thrust::raw_pointer_cast(X_d.data()), thrust::raw_pointer_cast(X_h), N * float_size, hipMemcpyHostToDevice));

    square<float> unary_op;
    thrust::plus<float> binary_op;

    float result = thrust::transform_reduce(X_d.begin(), X_d.end(), unary_op, 0.f, binary_op);

    return sqrt(result);
}

void norm_multi_vs_single(size_t N, int deviceCount)
{
    size_t float_size = sizeof(float);

    float* X_h = nullptr;
    checkCudaError(hipHostAlloc(&X_h, float_size * N, 0));
    thrust::tabulate(X_h, X_h + N, get_rand_number(43, 10));

    float result_single = norm_single(X_h, N, deviceCount);
    float result_multi = norm_multi(X_h, N, deviceCount);

    if (result_single  - result_multi > 1e-5)
    {
        std::cout << "right result" << std::endl;
    }
    else
    {
        std::cout << "single: " << result_single << std::endl;
        std::cout << "multi: " << result_multi << std::endl;
    }
}

int main(int argc, char **argv)
{ 
    int deviceCount;
    checkCudaError(hipGetDeviceCount(&deviceCount));

    //saxpy_multi_vs_single(100000000, deviceCount);
    norm_multi_vs_single(4, deviceCount);

    return 0;
}
