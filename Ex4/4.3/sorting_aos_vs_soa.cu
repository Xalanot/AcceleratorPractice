#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/random.h>
#include <assert.h>
#include <chrono>

// This examples compares sorting performance using Array of Structures (AoS)
// and Structure of Arrays (SoA) data layout.  Legacy applications will often
// store data in C/C++ structs, such as MyStruct defined below.  Although 
// Thrust can process array of structs, it is typically less efficient than
// the equivalent structure of arrays layout.  In this particular example,
// the optimized SoA approach is approximately *five times faster* than the
// traditional AoS method.  Therefore, it is almost always worthwhile to
// convert AoS data structures to SoA.

using Clock = std::chrono::high_resolution_clock;
using Duration = std::chrono::duration<double>;

struct MyStruct
{
  int key;
  float value;

  __host__ __device__
    bool operator<(const MyStruct other) const
    {
      return key < other.key;
    }
};

void initialize_keys(thrust::host_vector<int>& keys)
{
  thrust::default_random_engine rng;
  thrust::uniform_int_distribution<int> dist(0, 2147483647);

  for(size_t i = 0; i < keys.size(); i++)
    keys[i] = dist(rng);
}


void initialize_keys(thrust::host_vector<MyStruct>& structures)
{
  thrust::default_random_engine rng;
  thrust::uniform_int_distribution<int> dist(0, 2147483647);

  for(size_t i = 0; i < structures.size(); i++)
    structures[i].key = dist(rng);
}

void sortAoS(size_t N)
{
    thrust::host_vector<MyStruct> structures_h(N);
    initialize_keys(structures_h);

    hipDeviceSynchronize();

    auto start = Clock::now();

    thrust::device_vector<MyStruct> structures_d = structures_h;

    thrust::sort(structures_d.begin(), structures_d.end());

    structures_h = structures_d;

    hipDeviceSynchronize();

    auto end = Clock::now();

    auto duration = static_cast<Duration>(end - start);

    std::cout << "sortAoS: " << duration.count() << std::endl;

    assert(thrust::is_sorted(structures_h.begin(), structures_h.end()));
}

void sortSoA(size_t N)
{
    thrust::host_vector<int>   keys_h(N);
    thrust::host_vector<float> values_h(N);

    initialize_keys(keys_h);

    hipDeviceSynchronize();

    auto start = Clock::now();

    thrust::device_vector<int> keys_d = keys_h;
    thrust::device_vector<float> values_d = values_h;

    thrust::sort_by_key(keys_d.begin(), keys_d.end(), values_d.begin());

    keys_h = keys_d;
    values_h = values_d;

    hipDeviceSynchronize();

    auto end = Clock::now();

    auto duration = static_cast<Duration>(end - start);

    std::cout << "sortSoA: " << duration.count() << std::endl;

    assert(thrust::is_sorted(keys_h.begin(), keys_h.end()));
}

struct get_aos
{
  __host__ __device__
  MyStruct operator() (thrust::tuple<int, float> const& tuple)
  {
    MyStruct str;
    str.key = thrust::get<0>(tuple);
    str.value = thrust::get<1>(tuple);
    return str;
  }
};

struct get_soa
{
  __host__ __device__ 
  thrust::tuple<int, float> operator()(MyStruct const& str) 
  {
    return thrust::tuple<int, float> (str.key, str.value);
  }
};

void sort3(size_t N)
{
    thrust::host_vector<MyStruct> structures_h(N);
    thrust::device_vector<MyStruct> structures_d(N);
    thrust::device_vector<int> keys(N);
    thrust::device_vector<float> values(N);

    initialize_keys(structures_h);

    hipDeviceSynchronize();

    auto start = Clock::now();

    // Copy AoS to SoA on device
    /*auto transform_soa_begin = thrust::make_transform_iterator(structures_h.begin(), get_soa());
    auto transform_soa_end = thrust::make_transform_iterator(structures_h.end(), get_soa());
    thrust::copy(transform_soa_begin, transform_soa_end, thrust::make_zip_iterator(thrust::make_tuple(keys.begin(), values.begin())));*/
    structures_d = structures_h;
    thrust::transform(structures_d.begin(), structures_d.end(), keys.begin(), [] __device__ __host__ (MyStruct str) {return str.key;});
    thrust::transform(structures_d.begin(), structures_d.end(), values.begin(), [] __device__ __host__ (MyStruct str) {return str.value;});
    
    // Sort on the device with SoA format
    thrust::sort_by_key(keys.begin(), keys.end(), values.begin());
    
    auto transform_aos_begin = thrust::make_transform_iterator(thrust::make_zip_iterator(thrust::make_tuple(keys.begin(), values.begin())), get_aos());
    auto transform_aos_end = thrust::make_transform_iterator(thrust::make_zip_iterator(thrust::make_tuple(keys.end(), values.end())), get_aos());
    // Transfer data back to host
    thrust::copy(transform_aos_begin, transform_aos_end, structures_h.begin());
    
    hipDeviceSynchronize();

    auto end = Clock::now();

    auto duration = static_cast<Duration>(end - start);
    std::cout << "sort3: " << duration.count() << std::endl;
    assert(thrust::is_sorted(structures_h.begin(), structures_h.end()));
}

int main(void)
{
  size_t N = 2 * 1024 * 1024;

  // Sort Key-Value pairs using Array of Structures (AoS) storage 
  sortAoS(N);

  // Sort Key-Value pairs using Structure of Arrays (SoA) storage 
  sortSoA(N);

  // Sort 3
  sort3(N);

  return 0;
}

