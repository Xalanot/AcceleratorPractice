#include "hip/hip_runtime.h"
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/pair.h>
#include <thrust/random.h>
#include <thrust/extrema.h>

// This example shows how to compute a bounding box
// for a set of points in two dimensions.

struct point2d
{
  float x, y;
  
  __host__ __device__
  point2d() : x(0), y(0) {}
  
  __host__ __device__
  point2d(float _x, float _y) : x(_x), y(_y) {}
};

// bounding box type
struct bbox
{
  // construct an empty box
  __host__ __device__
  bbox() {}

  // construct a box from a single point
  __host__ __device__
  bbox(const point2d &point)
    : lower_left(point), upper_right(point)
  {}

  // construct a box from a single point
  __host__ __device__
  bbox& operator=(const point2d &point)
  {
    lower_left = point;
    upper_right = point;
    return *this;
  }

  // construct a box from a pair of points
  __host__ __device__
  bbox(const point2d &ll, const point2d &ur)
    : lower_left(ll), upper_right(ur)
  {}

  point2d lower_left, upper_right;
};

// reduce a pair of bounding boxes (a,b) to a bounding box containing intersection of a and b
struct bbox_reduction : public thrust::binary_function<bbox,bbox,bbox>
{
  __host__ __device__
  bbox operator()(bbox a, bbox b)
  {
    // lower left corner
    point2d ll(thrust::max(a.lower_left.x, b.lower_left.x), thrust::max(a.lower_left.y, b.lower_left.y));
    
    // upper right corner
    point2d ur(thrust::min(a.upper_right.x, b.upper_right.x), thrust::min(a.upper_right.y, b.upper_right.y));
    
    return bbox(ll, ur);
  }
};

int main(void)
{
  const size_t N = 40;
  thrust::default_random_engine rng;
  thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);
  
  // allocate storage for points
  thrust::device_vector<bbox> bboxes(N);
  
  // generate some random points in the unit square
  for(size_t i = 0; i < N; i++)
  {
      float x = u01(rng);
      float y = u01(rng);
      point2d ll(x,y);

      x = u01(rng);
      y = u01(rng);
      point2d ur(x,y);

      bboxes[i] = bbox(ll, ur);
  }
  
  // binary reduction operation
  bbox_reduction binary_op;
  
  // compute the intersection bounding box for the point set
  bbox result = thrust::reduce(bboxes.begin(), bboxes.end(), bboxes[0], binary_op);
  
  // print output
  std::cout << "intersection bounding box " << std::fixed;
  std::cout << "(" << result.lower_left.x  << "," << result.lower_left.y  << ") ";
  std::cout << "(" << result.upper_right.x << "," << result.upper_right.y << ")" << std::endl;
  
  return 0;
}
