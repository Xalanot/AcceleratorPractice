#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>

#include <chrono>
#include <iostream>
#include <vector>

#include "pml/csvwriter.h"
#include "pml/measurement.h"


struct saxpy_functor
{
    const float a;

    saxpy_functor(float _a) : a(_a) {}

    __host__ __device__
    float operator()(float const& x, float const& y) const
    {
        return a * x + y;
    }
};

struct is_mod10 : public thrust::unary_function<float, bool> {
	__host__ __device__ bool operator()(const float& x) const {
		return (int) x % 10 == 0;
	}
};

void saxpy_fast(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{
    thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), saxpy_functor(A));
}

void saxpy_fast(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y, thrust::device_vector<float>& Z)
{
    thrust::transform(X.begin(), X.end(), Y.begin(), Z.begin(), saxpy_functor(A));
}

void saxpyIf_fast(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{    
    thrust::transform_if(X.begin(), X.end(), Y.begin(), Y.begin(), Y.begin(), saxpy_functor(A), is_mod10());
}

int main(int argc, char** argv)
{
    float a = 2;

    std::vector<int> sizes;
    std::vector<Measurement<std::chrono::microseconds>> saxpyFastMeasurements;
    std::vector<Measurement<std::chrono::microseconds>> saxpyFast3Measurements;
    std::vector<Measurement<std::chrono::microseconds>> saxpyIf_FastMeasurements;

    for (int size = 1; size < 10e6; size *= 10)
    {
        sizes.push_back(size);

        // setup host vector
        thrust::host_vector<float> X_h(size, 1);
        thrust::host_vector<float> Y_h(size);
        thrust::sequence(Y_h.begin(), Y_h.end());

        // copy to device
        thrust::device_vector<float> X_d(X_h);
        thrust::device_vector<float> Y_d(Y_h);

        // saxpy_fast
        Measurement<std::chrono::microseconds> saxpyFastMeasurement;
        hipDeviceSynchronize();
        saxpyFastMeasurement.start();
        saxpy_fast(a, X_d, Y_d);
        hipDeviceSynchronize();
        saxpyFastMeasurement.stop();
        saxpyFastMeasurements.push_back(saxpyFastMeasurement);

        // saxpy_fast3
        Measurement<std::chrono::microseconds> saxpyFast3Measurement;
        thrust::device_vector<float> Z_d(size);
        hipDeviceSynchronize();
        saxpyFast3Measurement.start();
        saxpy_fast3(a, X_d, Y_d, Z_d);
        hipDeviceSynchronize();
        saxpyFast3Measurement.stop();
        saxpyFast3Measurements.push_back(saxpyFast3Measurement);

        // saxpyIF_fast
        Measurement<std::chrono::microseconds> saxpyIf_FastMeasurement;
        hipDeviceSynchronize();
        saxpyIf_FastMeasurement.start();
        saxpyIf_fast(a, X_d, Y_d);
        hipDeviceSynchronize();
        saxpyIf_FastMeasurement.stop();
        saxpyIf_FastMeasurements.push_back(saxpyIf_FastMeasurement);

        // copy to host
        X_h = X_d;
        Y_h = Y_d;
    }

    CSVWriter csvwriter("saxpy3.csv");
    std::vector<std::string> headerNames {"size", "saxpyFast", "saxpyFast3", "saxpyIf_Fast"};
    csvwriter.setHeaderNames(std::move(headerNames));
    csvwriter.write(sizes, saxpyFastMeasurements, saxpyFast3Measurements, saxpyIf_FastMeasurements);

    return 0;
}