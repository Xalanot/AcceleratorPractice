#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>

#include <chrono>
#include <iostream>
#include <vector>

#include "pml/csvwriter.h"
#include "pml/measurement.h"


struct saxpy_functor
{
    const float a;

    saxpy_functor(float _a) : a(_a) {}

    __host__ __device__
    float operator()(float const& x, float const& y) const
    {
        return a * x + y;
    }
};

struct is_mod10 : public thrust::unary_function<float, bool> {
	__host__ __device__ bool operator()(const float& x) const {
		return (int) x % 10 == 0;
	}
};

void saxpy_fast(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{
    thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), saxpy_functor(A));
}

void saxpy_fast3(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y, thrust::device_vector<float>& Z)
{
    thrust::transform(X.begin(), X.end(), Y.begin(), Z.begin(), saxpy_functor(A));
}

void saxpyIf_fast(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{    
    thrust::transform_if(X.begin(), X.end(), Y.begin(), Y.begin(), Y.begin(), saxpy_functor(A), is_mod10());
}

int main(int argc, char** argv)
{
    float a = 2;
    int iterations = 100;

    std::vector<int> sizes;
    std::vector<MeasurementSeries<std::chrono::microseconds>> saxpyFast;
    std::vector<MeasurementSeries<std::chrono::microseconds>> saxpyFast3;
    std::vector<MeasurementSeries<std::chrono::microseconds>> saxpyIf_Fast;

    for (int size = 1; size < 10e6; size *= 10)
    {
        sizes.push_back(size);

        // setup host vector
        thrust::host_vector<float> X_h(size, 1);
        thrust::host_vector<float> Y_h(size);
        thrust::sequence(Y_h.begin(), Y_h.end());

        // copy to device
        thrust::device_vector<float> X_d(X_h);
        thrust::device_vector<float> Y_d(Y_h);

        // saxpy_fast
        MeasurementSeries<std::chrono::microseconds> saxpyFastMeasurementSeries;
        for (int i = 0; i < iterations; ++i)
        {
            hipDeviceSynchronize();
            saxpyFastMeasurementSeries.start();
            saxpy_fast(a, X_d, Y_d);
            hipDeviceSynchronize();
            saxpyFastMeasurementSeries.stop();
        }
        saxpyFast.push_back(saxpyFastMeasurementSeries);

        // saxpy_fast3
        MeasurementSeries<std::chrono::microseconds> saxpyFast3MeasurementSeries;
        thrust::device_vector<float> Z_d(size);
        for (int i = 0; i < iterations; ++i)
        {
            hipDeviceSynchronize();
            saxpyFast3MeasurementSeries.start();
            saxpy_fast3(a, X_d, Y_d, Z_d);
            hipDeviceSynchronize();
            saxpyFast3MeasurementSeries.stop();
        }
        saxpyFast3.push_back(saxpyFast3MeasurementSeries);

        // saxpyIF_fast
        MeasurementSeries<std::chrono::microseconds> saxpyIf_FastMeasurementSeries;
        for (int i = 0; i < iterations; ++i)
        {
            hipDeviceSynchronize();
            saxpyIf_FastMeasurement.start();
            saxpyIf_fast(a, X_d, Y_d);
            hipDeviceSynchronize();
            saxpyIf_FastMeasurement.stop();
            saxpyIf_FastMeasurements.push_back(saxpyIf_FastMeasurement);
        }
        saxpyIf_Fast.push_back(saxpyIf_FastMeasurementSeries);

        // copy to host
        X_h = X_d;
        Y_h = Y_d;
    }

    CSVWriter csvwriter("saxpy3.csv");
    std::vector<std::string> headerNames {"size", "saxpyFast", "saxpyFast3", "saxpyIf_Fast"};
    csvwriter.setHeaderNames(std::move(headerNames));
    csvwriter.write(sizes, saxpyFast, saxpyFast3, saxpyIf_Fast);

    return 0;
}