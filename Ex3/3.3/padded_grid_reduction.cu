#include "hip/hip_runtime.h"
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/random.h>
#include <thrust/extrema.h>
#include <cmath>
#include <iomanip>
#include <float.h>

// This example computes the minimum and maximum values
// over a padded grid.  The padded values are not considered
// during the reduction operation.


// transform a tuple (int,value) into a tuple (bool,value,value)
// where the bool is true for valid grid values and false for 
// values in the padded region of the grid
template <typename IndexType, typename ValueType>
struct transform_tuple : 
    public thrust::unary_function< thrust::tuple<IndexType,ValueType>, 
                                   thrust::tuple<bool,ValueType,ValueType> >
{
  typedef typename thrust::tuple<IndexType,ValueType>      InputTuple;
  typedef typename thrust::tuple<bool,ValueType,ValueType> OutputTuple;

  IndexType n, N;

  transform_tuple(IndexType n, IndexType N) : n(n), N(N) {}

  __host__ __device__
    OutputTuple operator()(const InputTuple& t) const
    { 
      bool is_valid = (thrust::get<0>(t) % N) < n;
      return OutputTuple(is_valid, thrust::get<1>(t), thrust::get<1>(t));
    }
};


// reduce two tuples (bool,value,value) into a single tuple such that output
// contains the smallest and largest *valid* values.
template <typename IndexType, typename ValueType>
struct reduce_tuple :
    public thrust::binary_function< thrust::tuple<bool,ValueType,ValueType>,
                                    thrust::tuple<bool,ValueType,ValueType>,
                                    thrust::tuple<bool,ValueType,ValueType> >
{
  typedef typename thrust::tuple<bool,ValueType,ValueType> Tuple;

  __host__ __device__
    Tuple operator()(const Tuple& t0, const Tuple& t1) const
    { 
      if(thrust::get<0>(t0) && thrust::get<0>(t1)) // both valid
        return Tuple(true, 
            thrust::min(thrust::get<1>(t0), thrust::get<1>(t1)),
            thrust::max(thrust::get<2>(t0), thrust::get<2>(t1)));
      else if (thrust::get<0>(t0))
        return t0;
      else if (thrust::get<0>(t1))
        return t1;
      else
        return t1; // if neither is valid then it doesn't matter what we return
    }
};

template <typename ValueType>
struct reduce_tuple_new :
    public thrust::binary_function< thrust::tuple<ValueType,ValueType>,
                                    thrust::tuple<ValueType,ValueType>,
                                    thrust::tuple<ValueType,ValueType> >
{
  typedef typename thrust::tuple<ValueType,ValueType> Tuple;

  __host__ __device__
    Tuple operator()(const Tuple& t0, const Tuple& t1) const
    { 
      return Tuple(thrust::min(thrust::get<0>(t0), thrust::get<0>(t1)),
                   thrust::max(thrust::get<1>(t0), thrust::get<1>(t1)));
    }
};

thrust::device_vector<int> generateMap(int M, int n, int N)
{
    thrust::device_vector<int> map(M * n);
    int currentIndex = 0;
    for (int i = 0; i < M * N; ++i)
    {
        if ( (i % N) < n)
        {
            map[currentIndex] = i;
            currentIndex++;
        }
    }

    return map;
}


int main(void)
{
  int M = 3;  // number of rows
  int n = 4;  // number of columns excluding padding
  int N = 5;  // number of columns including padding

  thrust::default_random_engine rng(12345);
  thrust::uniform_real_distribution<float> dist(0.0f, 1.0f);

  thrust::device_vector<float> data(M * N, -1);

  // initialize valid values in grid
  for(int i = 0; i < M; i++)
    for(int j = 0; j < n; j++)
      data[i * N + j] = dist(rng);

  // print full grid
  std::cout << "padded grid" << std::endl;
  std::cout << std::fixed << std::setprecision(4);
  for(int i = 0; i < M; i++)
  {
    std::cout << " ";
    for(int j = 0; j < N; j++)
    {
      std::cout << data[i * N + j] << " ";
    }   
    std::cout << "\n";
  }
  std::cout << "\n";

  // compute min & max over valid region of the 2d grid
  typedef thrust::tuple<bool, float, float> result_type;

  result_type                 init(true, FLT_MAX, -FLT_MAX); // initial value
  transform_tuple<int,float>  unary_op(n, N);                // transformation operator
  reduce_tuple<int,float>     binary_op;                     // reduction operator

  result_type result = 
    thrust::transform_reduce(
        thrust::make_zip_iterator(thrust::make_tuple(thrust::counting_iterator<int>(0), data.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(thrust::counting_iterator<int>(0), data.begin())) + data.size(),
        unary_op,
        init,
        binary_op);

  std::cout << "minimum value: " << thrust::get<1>(result) << std::endl;
  std::cout << "maximum value: " << thrust::get<2>(result) << std::endl;

  // new
  std::cout << "new" << std::endl;
  thrust::device_vector<int> map = generateMap(M, n, N);
  auto zip_iterator_first = thrust::make_zip_iterator(thrust::make_tuple(data.begin(), data.begin()));
  auto zip_iterator_last = thrust::make_zip_iterator(thrust::make_tuple(data.end(), data.end()));
  auto permutation_iterator_first = (zip_iterator_first, map.begin());
  auto permutation_iterator_last = (zip_iterator_first, map.end());
  reduce_tuple_new<float, float> binary_op_new;
  result_type result_new = 
    thrust::reduce(
        permutation_iterator_first,
        permutation_iterator_last,
        init,
        binary_op_new);

  std::cout << "minimum value: " << thrust::get<1>(result_new) << std::endl;
  std::cout << "maximum value: " << thrust::get<2>(result_new) << std::endl;
    


  return 0;
}

